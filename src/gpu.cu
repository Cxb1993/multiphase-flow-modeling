#include "hip/hip_runtime.h"
#include "gpu.h"
#include "shared_test.cu"

__constant__ consts gpu_def [1];
__device__ ptr_Arrays DevArraysPtr[1];
extern double *HostBuffer;
extern double *DevBuffer;

#include "gauss.cu"

#include "three-phase.cu"

#ifdef ENERGY
#include "energy.cu"
#endif

__device__ void device_assign_ro(int local)
{
#ifdef ENERGY
	// !!! Вынести коэффициенты теплового расширения в const consts &def и использовать T_0 оттуда же
	double alfa_w = 1.32E-7; // 1/K !!! E-4
	double alfa_n = 9.2E-7;
	double T_0 = 273;

	DevArraysPtr->ro_w[local] = gpu_def->ro0_w * (1. + (gpu_def->beta_w) * (DevArraysPtr->P_w[local] - gpu_def->P_atm) - alfa_w * (DevArraysPtr->T[local] - T_0));
	DevArraysPtr->ro_n[local] = gpu_def->ro0_n * (1. + (gpu_def->beta_n) * (DevArraysPtr->P_n[local] - gpu_def->P_atm) - alfa_n * (DevArraysPtr->T[local] - T_0));
	DevArraysPtr->ro_g[local] = gpu_def->ro0_g * (DevArraysPtr->P_g[local] / gpu_def->P_atm) * (T_0 / DevArraysPtr->T[local]);
#else
	DevArraysPtr->ro_w[local] = gpu_def->ro0_w * (1. + (gpu_def->beta_w) * (DevArraysPtr->P_w[local] - gpu_def->P_atm));
	DevArraysPtr->ro_n[local] = gpu_def->ro0_n * (1. + (gpu_def->beta_n) * (DevArraysPtr->P_n[local] - gpu_def->P_atm));
	DevArraysPtr->ro_g[local] = gpu_def->ro0_g * DevArraysPtr->P_g[local] / gpu_def->P_atm;
#endif
	device_test_positive(DevArraysPtr->ro_g[local], __FILE__, __LINE__);
	device_test_positive(DevArraysPtr->ro_w[local], __FILE__, __LINE__);
	device_test_positive(DevArraysPtr->ro_n[local], __FILE__, __LINE__);
}

// Вычисление координаты точки, через которую будет вычисляться значение на границе (i1, j1, k1)
__device__ int device_set_boundary_basic_coordinate(int i, int j, int k)
{
	int i1, j1, k1;

	i1 = i;
	j1 = j;
	k1 = k;

	if (i == 0)
	{
		i1 ++;
	}
	if (i == (gpu_def->locNx) - 1)
	{
		i1 --;
	}
	if (j == 0)
	{
		j1 ++;
	}
	if (j == (gpu_def->locNy) - 1)
	{
		j1 --;
	}
	if ((k == 0) && ((gpu_def->locNz) > 2))
	{
		k1 ++;
	}
	if ((k == (gpu_def->locNz) - 1) && ((gpu_def->locNz) > 2))
	{
		k1 --;
	}

	return (i1 + j1 * (gpu_def->locNx) + k1 * (gpu_def->locNx) * (gpu_def->locNy));
}

// Расчет центральной разности
__device__ double central_difference (double* ptr, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return (*(ptr+1) - *(ptr-1) )/ (2. * (gpu_def->hx));	
		}
	case 'y':
		{
			return (*(ptr + gpu_def->locNx) - *(ptr - gpu_def->locNx) )/ (2. * (gpu_def->hy));
		}
	case 'z':
		{
			return (*(ptr + gpu_def->locNx * (gpu_def->locNy)) - *(ptr - gpu_def->locNx * (gpu_def->locNy)) )/ (2. * (gpu_def->hz));
		}
	default:
		{
			device_print_error("Axis of [central_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Расчет центральной разности для произведения двух элементов структуры
__device__ double multi_central_difference (double* ptr1, double* ptr2, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return ((*(ptr1+1)) * (*(ptr2+1)) - (*(ptr1-1)) * (*(ptr2-1)) )/ (2. * (gpu_def->hx));
		}
	case 'y':
		{
			return ((*(ptr1+gpu_def->locNx)) * (*(ptr2+gpu_def->locNx)) - (*(ptr1-gpu_def->locNx)) * (*(ptr2-gpu_def->locNx)) )/ (2. * (gpu_def->hy));
		}
	case 'z':
		{
			return ((*(ptr1+gpu_def->locNx * (gpu_def->locNy))) * (*(ptr2+gpu_def->locNx * (gpu_def->locNy)))
				- (*(ptr1-gpu_def->locNx * (gpu_def->locNy))) * (*(ptr2-gpu_def->locNx * (gpu_def->locNy))) )/ (2. * (gpu_def->hz));
		}
	default:
		{
			device_print_error("Axis of [central_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Расчет направленной разности
__device__ double directed_difference (double* P, double* Xi, double* ro, char axis)
{
	double x1 = 0, x2 = 0;
	switch (axis)
	{
	case 'x':
		{
			x2 = -device_right_difference (P, 'x');
			x1 = -device_left_difference (P, 'x');
			return (((x2 + fabs(x2)) / 2. - (x1 - fabs(x1)) / 2.) * (*Xi) * (*ro) -
				(x1 + fabs(x1)) / 2. * (*(Xi-1)) * (*(ro-1)) +
				(x2 - fabs(x2)) / 2. * (*(Xi+1)) * (*(ro+1))) / gpu_def->hx * (-1.0);
		}
	case 'y':
		{
			x2 = -device_right_difference (P, 'y') + gpu_def->g_const * (*ro);
			x1 = -device_left_difference (P, 'y') + gpu_def->g_const * (*ro);
			return (((x2 + fabs(x2)) / 2. - (x1 - fabs(x1)) / 2.) * (*Xi) * (*ro) -
				(x1 + fabs(x1)) / 2. * (*(Xi - gpu_def->locNx)) * (*(ro - gpu_def->locNx)) +
				(x2 - fabs(x2)) / 2. * (*(Xi + gpu_def->locNx)) * (*(ro + gpu_def->locNx))) / gpu_def->hy * (-1.0);
		}
	case 'z':
		{
			x2 = -device_right_difference (P, 'z');
			x1 = -device_left_difference (P, 'z');
			return (((x2 + fabs(x2)) / 2. - (x1 - fabs(x1)) / 2.) * (*Xi) * (*ro) -
				(x1 + fabs(x1)) / 2. * (*(Xi - gpu_def->locNx * (gpu_def->locNy))) * (*(ro - gpu_def->locNx * (gpu_def->locNy))) +
				(x2 - fabs(x2)) / 2. * (*(Xi + gpu_def->locNx * (gpu_def->locNy))) * (*(ro + gpu_def->locNx * (gpu_def->locNy)))) / gpu_def->hz * (-1.0);
		}
	default:
		{
			device_print_error("Axis of [directed_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Расчет левой разности
__device__ double device_left_difference (double* ptr, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return (*ptr - *(ptr-1) )/ gpu_def->hx;	
		}
	case 'y':
		{
			return (*ptr - *(ptr-gpu_def->locNx) )/ gpu_def->hy;
		}
	case 'z':
		{
			return (*ptr - *(ptr - gpu_def->locNx * (gpu_def->locNy)) )/ gpu_def->hz;
		}
	default:
		{
			device_print_error("Axis of [left_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Расчет правой разности
__device__ double device_right_difference (double* ptr, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return (*(ptr+1) - *ptr )/ gpu_def->hx;	
		}
	case 'y':
		{
			return (*(ptr + gpu_def->locNx) - *ptr )/ gpu_def->hy;
		}
	case 'z':
		{
			return (*(ptr + gpu_def->locNx * (gpu_def->locNy)) - *ptr )/ gpu_def->hz;
		}
	default:
		{
			device_print_error("Axis of [right_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Расчет divgrad для элемента структуры
__device__ double divgrad (double* ptr1, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return ((*(ptr1+1)) - 2 * (*ptr1) + (*(ptr1-1))) / ((gpu_def->hx) * (gpu_def->hx));
		}
	case 'y':
		{
			return ((*(ptr1+gpu_def->locNx)) - 2 * (*ptr1) + (*(ptr1-gpu_def->locNx))) / ((gpu_def->hy) * (gpu_def->hy));
		}
	case 'z':
		{
			return ((*(ptr1+gpu_def->locNx * (gpu_def->locNy))) - 2 * (*ptr1) + (*(ptr1-gpu_def->locNx * (gpu_def->locNy)))) / ((gpu_def->hz) * (gpu_def->hz));
		}
	default:
		{
			device_print_error("Axis of [right_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}


// Расчет divgrad для произведения двух элементов структуры
__device__ double multi_divgrad (double* ptr1, double* ptr2, char axis)
{
	switch (axis)
	{
	case 'x':
		{
			return ((*(ptr1+1)) * (*(ptr2+1)) - 2 * (*ptr1) * (*ptr2)
				+ (*(ptr1-1)) * (*(ptr2-1))) / ((gpu_def->hx) * (gpu_def->hx));
		}
	case 'y':
		{
			return ((*(ptr1+gpu_def->locNx)) * (*(ptr2+gpu_def->locNx)) - 2 * (*ptr1) * (*ptr2)
				+ (*(ptr1-gpu_def->locNx)) * (*(ptr2-gpu_def->locNx))) / ((gpu_def->hy) * (gpu_def->hy));
		}
	case 'z':
		{
			return ((*(ptr1+gpu_def->locNx * (gpu_def->locNy))) * (*(ptr2+gpu_def->locNx * (gpu_def->locNy))) - 2 * (*ptr1) * (*ptr2)
				+ (*(ptr1-gpu_def->locNx * (gpu_def->locNy))) * (*(ptr2-gpu_def->locNx * (gpu_def->locNy)))) / ((gpu_def->hz) * (gpu_def->hz));
		}
	default:
		{
			device_print_error("Axis of [right_difference] conversation is empty", __FILE__, __LINE__);
			return -1;
		}
	}
}

// Преобразование локальных координат процессора к глобальным
// Каждый процессор содержит дополнительную точку в массиве для
// обмена данными, если имеет соседа
// (если 2 соседа с обеих сторон,то +2 точки).
// Глобальные границы хранятся как обычные точки (отсюда и условие на rank==0)
__device__ int device_local_to_global(int local_index, char axis)
{
	int global_index = local_index;
	switch (axis)
	{
		case 'x':
		{
			global_index += gpu_def->rankx * (gpu_def->Nx) / gpu_def->sizex + min(gpu_def->rankx, gpu_def->Nx % gpu_def->sizex);
			break;
		}
		case 'y':
		{
			global_index += gpu_def->ranky * (gpu_def->Ny) / gpu_def->sizey + min(gpu_def->ranky, gpu_def->Ny % gpu_def->sizey);
			break;
		}
		case 'z':
		{
			global_index += gpu_def->rankz * (gpu_def->Nz) / gpu_def->sizez + min(gpu_def->rankz, gpu_def->Nz % gpu_def->sizez);
			break;
		}
		default:
		{
			//CUPRINTF("Error!");
		}
	}
	//some_test(global_index);
	return global_index;
}

// Функция вычисления "эффективной" плотности
__device__ double device_ro_eff_gdy(int local)
{
	double ro_g_dy = (DevArraysPtr->ro_g[local] * (1. - DevArraysPtr->S_w[local] - DevArraysPtr->S_n[local]) + DevArraysPtr->ro_w[local] * DevArraysPtr->S_w[local]
	                  + DevArraysPtr->ro_n[local] * DevArraysPtr->S_n[local]) * (DevArraysPtr->m[local]) * (gpu_def->g_const) * (gpu_def->hy);
	return ro_g_dy;
}

void prepare_all_vars()
{
	prepare_local_vars_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign P, Xi", __FILE__, __LINE__);
}

void solve_nonlinear_system()
{
	Newton_method_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign Pw and Sn", __FILE__, __LINE__);
}

// Расчет скорости в каждой точке сетки
__global__ void assign_u_kernel()
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (GPU_ACTIVE_POINT)
	{
		int local=i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy);
		if ((gpu_def->Nx) > 2)
		{
			if (i == 0)
			{
				DevArraysPtr->ux_w[local] = DevArraysPtr->Xi_w[local] * device_right_difference(DevArraysPtr->P_w+local, 'x');
				DevArraysPtr->ux_n[local] = DevArraysPtr->Xi_n[local] * device_right_difference(DevArraysPtr->P_n+local, 'x');
				DevArraysPtr->ux_g[local] = DevArraysPtr->Xi_g[local] * device_right_difference(DevArraysPtr->P_g+local, 'x');
			}
			else
			{
				if (i == (gpu_def->locNx) - 1)
				{
					DevArraysPtr->ux_w[local] = DevArraysPtr->Xi_w[local] * device_left_difference(DevArraysPtr->P_w+local, 'x');
					DevArraysPtr->ux_n[local] = DevArraysPtr->Xi_n[local] * device_left_difference(DevArraysPtr->P_n+local, 'x');
					DevArraysPtr->ux_g[local] = DevArraysPtr->Xi_g[local] * device_left_difference(DevArraysPtr->P_g+local, 'x');
				}
				else
				{
					DevArraysPtr->ux_w[local] = DevArraysPtr->Xi_w[local] * central_difference (DevArraysPtr->P_w+local, 'x');
					DevArraysPtr->ux_n[local] = DevArraysPtr->Xi_n[local] * central_difference (DevArraysPtr->P_n+local, 'x');
					DevArraysPtr->ux_g[local] = DevArraysPtr->Xi_g[local] * central_difference (DevArraysPtr->P_g+local, 'x');
				}
			}
		}
		else
		{
			DevArraysPtr->ux_w[local] = 0.;
			DevArraysPtr->ux_n[local] = 0.;
			DevArraysPtr->ux_g[local] = 0.;
		}

		if ((gpu_def->Ny) > 2)
		{
			if (j == 0)
			{
				DevArraysPtr->uy_w[local] = DevArraysPtr->Xi_w[local] * (device_right_difference (DevArraysPtr->P_w+local, 'y') - DevArraysPtr->ro_w[local] * (gpu_def->g_const));
				DevArraysPtr->uy_n[local] = DevArraysPtr->Xi_n[local] * (device_right_difference (DevArraysPtr->P_n+local, 'y') - DevArraysPtr->ro_n[local] * (gpu_def->g_const));
				DevArraysPtr->uy_g[local] = DevArraysPtr->Xi_g[local] * (device_right_difference (DevArraysPtr->P_g+local, 'y') - DevArraysPtr->ro_g[local] * (gpu_def->g_const));
			}
			else
			{
				if (j == (gpu_def->locNy) - 1)
				{
					DevArraysPtr->uy_w[local] = DevArraysPtr->Xi_w[local] * (device_left_difference (DevArraysPtr->P_w+local, 'y') - DevArraysPtr->ro_w[local] * (gpu_def->g_const));
					DevArraysPtr->uy_n[local] = DevArraysPtr->Xi_n[local] * (device_left_difference (DevArraysPtr->P_n+local, 'y') - DevArraysPtr->ro_n[local] * (gpu_def->g_const));
					DevArraysPtr->uy_g[local] = DevArraysPtr->Xi_g[local] * (device_left_difference (DevArraysPtr->P_g+local, 'y') - DevArraysPtr->ro_g[local] * (gpu_def->g_const));
				}
				else
				{
					DevArraysPtr->uy_w[local] = DevArraysPtr->Xi_w[local] * (central_difference (DevArraysPtr->P_w+local, 'y')	- DevArraysPtr->ro_w[local] * (gpu_def->g_const));
					DevArraysPtr->uy_n[local] = DevArraysPtr->Xi_n[local] * (central_difference (DevArraysPtr->P_n+local, 'y')	- DevArraysPtr->ro_n[local] * (gpu_def->g_const));
					DevArraysPtr->uy_g[local] = DevArraysPtr->Xi_g[local] * (central_difference (DevArraysPtr->P_g+local, 'y')	- DevArraysPtr->ro_g[local] * (gpu_def->g_const));
				}
			}
		}
		else
		{
			DevArraysPtr->uy_w[local] = 0.;
			DevArraysPtr->uy_n[local] = 0.;
			DevArraysPtr->uy_g[local] = 0.;
		}

		if ((gpu_def->Nz) > 2)
		{
			if (k == 0)
			{
				DevArraysPtr->uz_w[local] = DevArraysPtr->Xi_w[local] * device_right_difference (DevArraysPtr->P_w+local, 'z');
				DevArraysPtr->uz_n[local] = DevArraysPtr->Xi_n[local] * device_right_difference (DevArraysPtr->P_n+local, 'z');
				DevArraysPtr->uz_g[local] = DevArraysPtr->Xi_g[local] * device_right_difference (DevArraysPtr->P_g+local, 'z');
			}
			else
			{
				if (k == (gpu_def->locNz) - 1)
				{
					DevArraysPtr->uz_w[local] = DevArraysPtr->Xi_w[local] * device_left_difference (DevArraysPtr->P_w+local, 'z');
					DevArraysPtr->uz_n[local] = DevArraysPtr->Xi_n[local] * device_left_difference (DevArraysPtr->P_n+local, 'z');
					DevArraysPtr->uz_g[local] = DevArraysPtr->Xi_g[local] * device_left_difference (DevArraysPtr->P_g+local, 'z');
				}
				else
				{
					DevArraysPtr->uz_w[local] = DevArraysPtr->Xi_w[local] * central_difference (DevArraysPtr->P_w+local, 'z');
					DevArraysPtr->uz_n[local] = DevArraysPtr->Xi_n[local] * central_difference (DevArraysPtr->P_n+local, 'z');
					DevArraysPtr->uz_g[local] = DevArraysPtr->Xi_g[local] * central_difference (DevArraysPtr->P_g+local, 'z');
				}
			}
		}
		else
		{
			DevArraysPtr->uz_w[local] = 0.;
			DevArraysPtr->uz_n[local] = 0.;
			DevArraysPtr->uz_g[local] = 0.;
		}

		device_test_u(DevArraysPtr->ux_w[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->ux_n[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uy_w[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uy_n[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uz_w[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uz_n[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->ux_g[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uy_g[local], __FILE__, __LINE__);
		device_test_u(DevArraysPtr->uz_g[local], __FILE__, __LINE__);
	}
}

// Расчет скоростей во всех точках сетки
void u_calculation()
{
	assign_u_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign u", __FILE__, __LINE__);
}

// Расчет вспомогательной насыщенности в каждой точке сетки
__device__ void device_assign_S(int local)
{
	DevArraysPtr->S_g[local] = 1. - DevArraysPtr->S_w[local] - DevArraysPtr->S_n[local];
}

// Расчет ro*S в каждой точке сетки методом направленных разностей
__global__ void assign_roS_kernel_nr(double t)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (GPU_INTERNAL_POINT)
	{
		int local = i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy);

		if(! DevArraysPtr->m[local])
			return;

		double q_w = 0., q_n = 0., q_g = 0.;

		// Значения q на скважинах
		device_wells_q(i, j, k, &q_w, &q_n, &q_g);

		DevArraysPtr->roS_w[local] = DevArraysPtr->ro_w[local] * DevArraysPtr->S_w[local];
		DevArraysPtr->roS_g[local] = DevArraysPtr->ro_g[local]
		        * (1. - DevArraysPtr->S_w[local] - DevArraysPtr->S_n[local]);
		DevArraysPtr->roS_n[local] = DevArraysPtr->ro_n[local] * DevArraysPtr->S_n[local];

		double f_w = 0., f_n = 0., f_g = 0., A1 = 0., A2 = 0., A3 = 0.;

		if ((gpu_def->Nx) > 2)
		{
			f_w += directed_difference (DevArraysPtr->P_w+local, DevArraysPtr->Xi_w+local, DevArraysPtr->ro_w+local, 'x');
			f_n += directed_difference (DevArraysPtr->P_n+local, DevArraysPtr->Xi_n+local, DevArraysPtr->ro_n+local, 'x');
			f_g += directed_difference (DevArraysPtr->P_g+local, DevArraysPtr->Xi_g+local, DevArraysPtr->ro_g+local, 'x');

		}
		if ((gpu_def->Ny) > 2)
		{
			f_w += directed_difference (DevArraysPtr->P_w+local, DevArraysPtr->Xi_w+local, DevArraysPtr->ro_w+local, 'y');
			f_n += directed_difference (DevArraysPtr->P_n+local, DevArraysPtr->Xi_n+local, DevArraysPtr->ro_n+local, 'y');
			f_g += directed_difference (DevArraysPtr->P_g+local, DevArraysPtr->Xi_g+local, DevArraysPtr->ro_g+local, 'y');
		}
		if ((gpu_def->Nz) > 2)
		{
			f_w += directed_difference (DevArraysPtr->P_w+local, DevArraysPtr->Xi_w+local, DevArraysPtr->ro_w+local, 'z');
			f_n += directed_difference (DevArraysPtr->P_n+local, DevArraysPtr->Xi_n+local, DevArraysPtr->ro_n+local, 'z');
			f_g += directed_difference (DevArraysPtr->P_g+local, DevArraysPtr->Xi_g+local, DevArraysPtr->ro_g+local, 'z');

		}

		A1 = DevArraysPtr->roS_w[local] + (gpu_def->dt / DevArraysPtr->m[local]) * (q_w - f_w);
		A2 = DevArraysPtr->roS_n[local] + (gpu_def->dt / DevArraysPtr->m[local]) * (q_n - f_n);
		A3 = DevArraysPtr->roS_g[local] + (gpu_def->dt / DevArraysPtr->m[local]) * (q_g - f_g);

		DevArraysPtr->roS_w_old[local] = DevArraysPtr->roS_w[local];
		DevArraysPtr->roS_n_old[local] = DevArraysPtr->roS_n[local];
		DevArraysPtr->roS_g_old[local] = DevArraysPtr->roS_g[local];
		DevArraysPtr->roS_w[local] = A1;
		DevArraysPtr->roS_n[local] = A2;
		DevArraysPtr->roS_g[local] = A3;

		device_test_positive(DevArraysPtr->roS_w[local], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr->roS_n[local], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr->roS_g[local], __FILE__, __LINE__);
	}
}

// Расчет ro*S в каждой точке сетки
__global__ void assign_roS_kernel(double t)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (GPU_INTERNAL_POINT)
	{
		int local = i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy);
		double divgrad_w = 0, divgrad_n = 0, divgrad_g = 0, A1 = 0, A2 = 0, A3 = 0;
		double f_w = 0, f_n = 0, f_g = 0;

		DevArraysPtr->roS_w[local] = DevArraysPtr->ro_w[local] * DevArraysPtr->S_w[local];
		DevArraysPtr->roS_n[local] = DevArraysPtr->ro_n[local] * DevArraysPtr->S_n[local];
		DevArraysPtr->roS_g[local] = DevArraysPtr->ro_g[local] * DevArraysPtr->S_g[local];

		if ((gpu_def->Nx) > 2)
		{
			divgrad_w += multi_divgrad (DevArraysPtr->ro_w + local, DevArraysPtr->S_w + local, 'x');
			divgrad_n += multi_divgrad (DevArraysPtr->ro_n + local, DevArraysPtr->S_n + local, 'x');
			divgrad_g += multi_divgrad (DevArraysPtr->ro_g + local, DevArraysPtr->S_g + local, 'x');

/*			f_w += multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->uz_w + local, 'x');
			f_n += multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->uz_n + local, 'x');
			f_g += multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->uz_g + local, 'x');
*/
			f_w += (DevArraysPtr->ro_w[local] * DevArraysPtr->Xi_w[local] * divgrad(DevArraysPtr->P_w + local, 'x')
					+ central_difference (DevArraysPtr->P_w + local, 'x')
					* multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->Xi_w + local, 'x'));
			f_n += (DevArraysPtr->ro_n[local] * DevArraysPtr->Xi_n[local] * divgrad(DevArraysPtr->P_n + local, 'x')
					+ central_difference (DevArraysPtr->P_n + local, 'x')
					* multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->Xi_n + local, 'x'));
			f_g += (DevArraysPtr->ro_g[local] * DevArraysPtr->Xi_g[local] * divgrad(DevArraysPtr->P_g + local, 'x')
					+ central_difference (DevArraysPtr->P_g + local, 'x')
					* multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->Xi_g + local, 'x'));
		}
		if ((gpu_def->Ny) > 2)
		{
			divgrad_w += multi_divgrad (DevArraysPtr->ro_w + local, DevArraysPtr->S_w + local, 'y');
			divgrad_n += multi_divgrad (DevArraysPtr->ro_n + local, DevArraysPtr->S_n + local, 'y');
			divgrad_g += multi_divgrad (DevArraysPtr->ro_g + local, DevArraysPtr->S_g + local, 'y');

/*			f_w += multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->uz_w + local, 'y');
			f_n += multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->uz_n + local, 'y');
			f_g += multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->uz_g + local, 'y');
*/
			f_w += (DevArraysPtr->ro_w[local] * DevArraysPtr->Xi_w[local] * divgrad(DevArraysPtr->P_w + local, 'y')
					+ central_difference (DevArraysPtr->P_w + local, 'y')
					* multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->Xi_w + local, 'y')
					- (DevArraysPtr->ro_w[local + gpu_def->locNx] * DevArraysPtr->ro_w[local + gpu_def->locNx] * DevArraysPtr->Xi_w[local + gpu_def->locNx]
					   - DevArraysPtr->ro_w[local - gpu_def->locNx] * DevArraysPtr->ro_w[local - gpu_def->locNx] * DevArraysPtr->Xi_w[local - gpu_def->locNx])
					   / (2.0 * gpu_def->hy) * (gpu_def->g_const));
			f_n += (DevArraysPtr->ro_n[local] * DevArraysPtr->Xi_n[local] * divgrad(DevArraysPtr->P_n + local, 'y')
					+ central_difference (DevArraysPtr->P_n + local, 'y')
					* multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->Xi_n + local, 'y')
					- (DevArraysPtr->ro_n[local + gpu_def->locNx] * DevArraysPtr->ro_n[local + gpu_def->locNx] * DevArraysPtr->Xi_n[local + gpu_def->locNx]
					   - DevArraysPtr->ro_n[local - gpu_def->locNx] * DevArraysPtr->ro_n[local - gpu_def->locNx] * DevArraysPtr->Xi_n[local - gpu_def->locNx])
					   / (2.0 * gpu_def->hy) * (gpu_def->g_const));
			f_g += (DevArraysPtr->ro_g[local] * DevArraysPtr->Xi_g[local] * divgrad(DevArraysPtr->P_g + local, 'y')
					+ central_difference (DevArraysPtr->P_g + local, 'y')
					* multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->Xi_g + local, 'y')
					- (DevArraysPtr->ro_g[local + gpu_def->locNx] * DevArraysPtr->ro_g[local + gpu_def->locNx] * DevArraysPtr->Xi_g[local + gpu_def->locNx]
					   - DevArraysPtr->ro_g[local - gpu_def->locNx] * DevArraysPtr->ro_g[local - gpu_def->locNx] * DevArraysPtr->Xi_g[local - gpu_def->locNx])
					   / (2.0 * gpu_def->hy) * (gpu_def->g_const));
		}
		if ((gpu_def->Nz) > 2)
		{
			divgrad_w += multi_divgrad (DevArraysPtr->ro_w + local, DevArraysPtr->S_w + local, 'z');
			divgrad_n += multi_divgrad (DevArraysPtr->ro_n + local, DevArraysPtr->S_n + local, 'z');
			divgrad_g += multi_divgrad (DevArraysPtr->ro_g + local, DevArraysPtr->S_g + local, 'z');

/*			f_w += multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->uz_w + local, 'z');
			f_n += multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->uz_n + local, 'z');
			f_g += multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->uz_g + local, 'z');
*/
			f_w += (DevArraysPtr->ro_w[local] * DevArraysPtr->Xi_w[local] * divgrad(DevArraysPtr->P_w + local, 'z')
					+ central_difference (DevArraysPtr->P_w + local, 'z')
					* multi_central_difference (DevArraysPtr->ro_w + local, DevArraysPtr->Xi_w + local, 'z'));
			f_n += (DevArraysPtr->ro_n[local] * DevArraysPtr->Xi_n[local] * divgrad(DevArraysPtr->P_n + local, 'z')
					+ central_difference (DevArraysPtr->P_n + local, 'z')
					* multi_central_difference (DevArraysPtr->ro_n + local, DevArraysPtr->Xi_n + local, 'z'));
			f_g += (DevArraysPtr->ro_g[local] * DevArraysPtr->Xi_g[local] * divgrad(DevArraysPtr->P_g + local, 'z')
					+ central_difference (DevArraysPtr->P_g + local, 'z')
					* multi_central_difference (DevArraysPtr->ro_g + local, DevArraysPtr->Xi_g + local, 'z'));
		}

		divgrad_w *= DevArraysPtr->m[local] * (gpu_def->l) * (gpu_def->c_w);
		divgrad_n *= DevArraysPtr->m[local] * (gpu_def->l) * (gpu_def->c_n);
		divgrad_g *= DevArraysPtr->m[local] * (gpu_def->l) * (gpu_def->c_g);

		device_test_arrowhead(f_w, divgrad_w, __FILE__, __LINE__);
		device_test_arrowhead(f_n, divgrad_n, __FILE__, __LINE__);
		device_test_arrowhead(f_g, divgrad_g, __FILE__, __LINE__);

		double q_w = 0., q_n = 0., q_g = 0.;

		// Значения q на скважинах
		device_wells_q(i, j, k, &q_w, &q_n, &q_g);

		if ((t < 2 * (gpu_def->dt)) || TWO_LAYERS)
		{
			A1 = DevArraysPtr->roS_w[local] + ((gpu_def->dt) / DevArraysPtr->m[local]) * (q_w + divgrad_w - f_w);
			A2 = DevArraysPtr->roS_n[local] + ((gpu_def->dt) / DevArraysPtr->m[local]) * (q_n + divgrad_n - f_n);
			A3 = DevArraysPtr->roS_g[local] + ((gpu_def->dt) / DevArraysPtr->m[local]) * (q_g + divgrad_g - f_g);
		}
		else
		{
			A1 = (1. / ((DevArraysPtr->m[local]) * (gpu_def->dt) + 2. * (gpu_def->tau))) * (2. * (gpu_def->dt) * (gpu_def->dt) * (q_w + divgrad_w - f_w)
			        + ((DevArraysPtr->m[local]) * (gpu_def->dt) - 2. * (gpu_def->tau)) * DevArraysPtr->roS_w_old[local]
			        + 4. * (gpu_def->tau) * DevArraysPtr->roS_w[i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy)]);
			A2 = (1. / ((DevArraysPtr->m[local]) * (gpu_def->dt) + 2. * (gpu_def->tau))) * (2. * (gpu_def->dt) * (gpu_def->dt) * (q_n + divgrad_n - f_n)
			        + ((DevArraysPtr->m[local]) * (gpu_def->dt) - 2. * (gpu_def->tau)) * DevArraysPtr->roS_n_old[local]
			        + 4. * (gpu_def->tau) * DevArraysPtr->roS_n[i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy)]);

			A3 = (1. / ((DevArraysPtr->m[local]) * (gpu_def->dt) + 2. * (gpu_def->tau))) * (2. * (gpu_def->dt) * (gpu_def->dt) * (q_g + divgrad_g - f_g)
			        + ((DevArraysPtr->m[local]) * (gpu_def->dt) - 2. * (gpu_def->tau)) * DevArraysPtr->roS_g_old[local]
			        + 4. * (gpu_def->tau) * DevArraysPtr->roS_g[i + j * (gpu_def->locNx) + k * (gpu_def->locNx) * (gpu_def->locNy)]);
		}

		DevArraysPtr->roS_w_old[local] = DevArraysPtr->roS_w[local];
		DevArraysPtr->roS_n_old[local] = DevArraysPtr->roS_n[local];
		DevArraysPtr->roS_g_old[local] = DevArraysPtr->roS_g[local];
		DevArraysPtr->roS_w[local] = A1;
		DevArraysPtr->roS_n[local] = A2;
		DevArraysPtr->roS_g[local] = A3;

		device_test_positive(DevArraysPtr->roS_w[local], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr->roS_n[local], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr->roS_g[local], __FILE__, __LINE__);
	}
}

void find_values_from_partial_equations(double t)
{
#ifdef NR
	assign_roS_kernel_nr <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>(t);
#else
	assign_roS_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>(t);
#endif
	checkErrors("assign roS", __FILE__, __LINE__);
#ifdef ENERGY
	assign_E_new_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign E new", __FILE__, __LINE__);
#endif
}

// Применение граничных условий
void boundary_conditions()
{
#ifdef ENERGY
	Border_T_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign T", __FILE__, __LINE__);
#endif

	Border_S_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign S", __FILE__, __LINE__);

	Border_P_kernel <<< dim3(def.blocksX, def.blocksY, def.blocksZ), dim3(BlockNX, BlockNY, BlockNZ)>>>();
	checkErrors("assign Pw", __FILE__, __LINE__);
}

// Функция загрузки данных в память хоста
void load_data_to_host(double* HostArrayPtr, double* DevArrayPtr)
{
	hipMemcpy(HostArrayPtr, DevArrayPtr, (def.locNx) * (def.locNy) * (def.locNz)*sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

// Функция загрузки данных типа double в память ускорителя
void load_data_to_device(double* HostArrayPtr, double* DevArrayPtr)
{
	hipMemcpy(DevArrayPtr, HostArrayPtr, (def.locNx) * (def.locNy) * (def.locNz)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy double data to device", __FILE__, __LINE__);
}

// Функция загрузки данных типа int в память ускорителя
void load_data_to_device_int(int* HostArrayPtr, int* DevArrayPtr)
{
	hipMemcpy(DevArrayPtr, HostArrayPtr, (def.locNx) * (def.locNy) * (def.locNz)*sizeof(int), hipMemcpyHostToDevice);
	checkErrors("copy int data to device", __FILE__, __LINE__);
}

// Выделение памяти ускорителя под массив точек расчетной области
void device_memory_allocation()
{
	int buffer_size = 0;

	if(def.sizex > 1)
		buffer_size = (def.locNy) * (def.locNz);
	if(def.sizey > 1 && (def.locNx) * (def.locNz) > buffer_size)
		buffer_size = (def.locNx) * (def.locNz);
	if(def.sizez > 1 && (def.locNx) * (def.locNy) > buffer_size)
		buffer_size = (def.locNx) * (def.locNy);

	if(buffer_size) {
		hipMalloc((void**) &DevBuffer, buffer_size * sizeof(double));
	}

	int sz = (def.locNx) * (def.locNy) * (def.locNz) * sizeof(double);

	hipMalloc((void**) & (DevArraysPtrLoc->P_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->P_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->S_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ro_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ro_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ux_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uy_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uz_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ux_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uy_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uz_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->Xi_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->Xi_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_w_old), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_n_old), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->m), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->K), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->S_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->P_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->S_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ro_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->ux_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uy_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->uz_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->Xi_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->roS_g_old), sz);
#ifdef ENERGY
	hipMalloc((void**) & (DevArraysPtrLoc->T), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->H_w), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->H_n), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->H_g), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->H_r), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->E), sz);
	hipMalloc((void**) & (DevArraysPtrLoc->E_new), sz);
#endif
	ptr_Arrays *DevArraysTmp = new ptr_Arrays[1];
	DevArraysTmp[0] = *DevArraysPtrLoc;
	hipMemcpyToSymbol(HIP_SYMBOL(DevArraysPtr), DevArraysTmp, sizeof(ptr_Arrays));

	hipMemcpy((double*)(DevArraysPtrLoc->P_w), HostArraysPtr.P_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->P_n), HostArraysPtr.P_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->S_n), HostArraysPtr.S_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ro_w), HostArraysPtr.ro_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ro_n), HostArraysPtr.ro_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ux_w), HostArraysPtr.ux_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uy_w), HostArraysPtr.uy_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uz_w), HostArraysPtr.uz_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ux_n), HostArraysPtr.ux_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uy_n), HostArraysPtr.uy_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uz_n), HostArraysPtr.uz_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->Xi_w), HostArraysPtr.Xi_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->Xi_n), HostArraysPtr.Xi_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_w), HostArraysPtr.roS_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_n), HostArraysPtr.roS_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_w_old), HostArraysPtr.roS_w_old, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_n_old), HostArraysPtr.roS_n_old, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->m), HostArraysPtr.m, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->K), HostArraysPtr.K, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->S_w), HostArraysPtr.S_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->P_g), HostArraysPtr.P_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->S_g), HostArraysPtr.S_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ro_g), HostArraysPtr.ro_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->ux_g), HostArraysPtr.ux_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uy_g), HostArraysPtr.uy_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->uz_g), HostArraysPtr.uz_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->Xi_g), HostArraysPtr.Xi_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_g), HostArraysPtr.roS_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->roS_g_old), HostArraysPtr.roS_g_old, sz, hipMemcpyHostToDevice);
#ifdef ENERGY
	hipMemcpy((double*)(DevArraysPtrLoc->T), HostArraysPtr.T, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->H_w), HostArraysPtr.H_w, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->H_n), HostArraysPtr.H_n, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->H_g), HostArraysPtr.H_g, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->H_r), HostArraysPtr.H_r, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->E), HostArraysPtr.E, sz, hipMemcpyHostToDevice);
	hipMemcpy((double*)(DevArraysPtrLoc->E_new), HostArraysPtr.E_new, sz, hipMemcpyHostToDevice);
#endif

	checkErrors("memory allocation", __FILE__, __LINE__);
}

// Освобожение памяти ускорителя из под массива точек расчетной области
void device_memory_free()
{
	hipFree(DevBuffer);
	hipFree((double*)(DevArraysPtrLoc->P_w));
	hipFree((double*)(DevArraysPtrLoc->P_n));
	hipFree((double*)(DevArraysPtrLoc->S_n));
	hipFree((double*)(DevArraysPtrLoc->ro_w));
	hipFree((double*)(DevArraysPtrLoc->ro_n));
	hipFree((double*)(DevArraysPtrLoc->ux_w));
	hipFree((double*)(DevArraysPtrLoc->uy_w));
	hipFree((double*)(DevArraysPtrLoc->uz_w));
	hipFree((double*)(DevArraysPtrLoc->ux_n));
	hipFree((double*)(DevArraysPtrLoc->uy_n));
	hipFree((double*)(DevArraysPtrLoc->uz_n));
	hipFree((double*)(DevArraysPtrLoc->Xi_w));
	hipFree((double*)(DevArraysPtrLoc->Xi_n));
	hipFree((double*)(DevArraysPtrLoc->roS_w));
	hipFree((double*)(DevArraysPtrLoc->roS_n));
	hipFree((double*)(DevArraysPtrLoc->roS_w_old));
	hipFree((double*)(DevArraysPtrLoc->roS_n_old));
	hipFree((double*)(DevArraysPtrLoc->m));
	hipFree((double*)(DevArraysPtrLoc->K));
	hipFree((double*)(DevArraysPtrLoc->S_w));
	hipFree((double*)(DevArraysPtrLoc->P_g));
	hipFree((double*)(DevArraysPtrLoc->S_g));
	hipFree((double*)(DevArraysPtrLoc->ro_g));
	hipFree((double*)(DevArraysPtrLoc->ux_g));
	hipFree((double*)(DevArraysPtrLoc->uy_g));
	hipFree((double*)(DevArraysPtrLoc->uz_g));
	hipFree((double*)(DevArraysPtrLoc->Xi_g));
	hipFree((double*)(DevArraysPtrLoc->roS_g));
	hipFree((double*)(DevArraysPtrLoc->roS_g_old));
#ifdef ENERGY
	hipFree((double*)(DevArraysPtrLoc->T));
	hipFree((double*)(DevArraysPtrLoc->H_w));
	hipFree((double*)(DevArraysPtrLoc->H_n));
	hipFree((double*)(DevArraysPtrLoc->H_g));
	hipFree((double*)(DevArraysPtrLoc->H_r));
	hipFree((double*)(DevArraysPtrLoc->E));
	hipFree((double*)(DevArraysPtrLoc->E_new));
#endif
	checkErrors("memory release", __FILE__, __LINE__);
}

// Инициализация ускорителя
// Расчет происходит на ускорителе, номер которого равен
// номеру запускающего процессора
void device_initialization()
{
	// Было бы очень неплохо вместо GPU_PER_NODE использовать hipGetDeviceCount
	//int deviceCount;
	//hipGetDeviceCount ( &deviceCount );

	// Считаем, что ядер на узле не меньше, чем ускорителей
	int device = def.rank % GPU_PER_NODE;
	hipSetDevice(device);

	// Количество запускаемых блоков
	// Если число точек сетки не кратно размеру блока,
	// то количество блоков будет на 1 больше.
	def.blocksX = def.locNx / BlockNX;
	if ((def.locNx % BlockNX) != 0)
	{
		(def.blocksX)++;
	}
	def.blocksY = (def.locNy) / BlockNY;
	if ((def.locNy) % BlockNY != 0)
	{
		(def.blocksY)++;
	}
	def.blocksZ = (def.locNz) / BlockNZ;
	if ((def.locNz) % BlockNZ != 0)
	{
		(def.blocksZ)++;
	}

	consts* deff = new consts[1];
	deff[0] = def;
	hipMemcpyToSymbol(HIP_SYMBOL(gpu_def), deff, sizeof(consts));
	checkErrors("constant memory copy", __FILE__, __LINE__);

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, device);

	if (devProp.major < 2)
	{
		printf("\nError! Compute capability < 2, rank=%d\n", def.rank);
	}

	if (!def.rank)
	{
		//printf ( "Device %d\n", device );
		printf("Name : %s\n", devProp.name);
		printf("Compute capability : %d.%d\n", devProp.major, devProp.minor);
		printf("Total Global Memory : %ld\n", devProp.totalGlobalMem);
		printf("Shared memory per block: %ld\n", devProp.sharedMemPerBlock);
		printf("Registers per block : %d\n", devProp.regsPerBlock);
		printf("Warp size : %d\n", devProp.warpSize);
		printf("Max threads per block : %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory : %ld\n", devProp.totalConstMem);
		printf("Number of multiprocessors: %d\n",  devProp.multiProcessorCount);
		//printf("Kernel execution timeout: %s\n\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
		for (int i = 0; i < 3; ++i)
		{
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		}
		for (int i = 0; i < 3; ++i)
		{
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
		}


		// Максимальный размер расчетной сетки для ускорителя
		// sizeof(ptr_Arrays)/4 - количество параметров в точке, т.к. 4 -размер одного указателя
		printf("\nTotal NAPL_Filtration grid size : %ld\n\n", devProp.totalGlobalMem / (sizeof(ptr_Arrays)*sizeof(double) / 4));
	}

	// (def.locNx)+2 потому что 2NyNz на буфер обмена выделяется
	// Нужно переписать!!! Учесть размер буфера правильно!!!
	if ((def.locNx + 2) * (def.locNy) * (def.locNz) > (devProp.totalGlobalMem / (sizeof(ptr_Arrays)*sizeof(double) / 4)))
	{
		printf("\nError! Not enough memory at GPU, rank=%d\n", def.rank);
	}
	fflush(stdout);
}

// Финализация ускорителя
void device_finalization(void)
{
}

__global__ void load_exchange_data_part_xl_kernel(double* DevArray, double* DevBuffer)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (j < gpu_def->locNy && k < (gpu_def->locNz))
	{
		DevBuffer[j + (gpu_def->locNy)*k] = DevArray[1 + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k];
		device_test_nan(DevBuffer[j + (gpu_def->locNy)*k], __FILE__, __LINE__);
	}
}

__global__ void load_exchange_data_part_xr_kernel(double* DevArray, double* DevBuffer)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (j < gpu_def->locNy && k < (gpu_def->locNz))
	{
		DevBuffer[j + (gpu_def->locNy)*k] = DevArray[(gpu_def->locNx) - 2 + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k];
		device_test_nan(DevBuffer[j + (gpu_def->locNy)*k], __FILE__, __LINE__);
	}
}

__global__ void load_exchange_data_part_yl_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && k < (gpu_def->locNz))
	{
		DevBuffer[i + (gpu_def->locNx)*k] = DevArray[i + (gpu_def->locNx) + (gpu_def->locNx) * (gpu_def->locNy) * k];
		device_test_nan(DevBuffer[i + (gpu_def->locNx)*k], __FILE__, __LINE__);
	}
}

__global__ void load_exchange_data_part_yr_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && k < (gpu_def->locNz))
	{
		DevBuffer[i + (gpu_def->locNx)*k] = DevArray[i + (gpu_def->locNx) * (gpu_def->locNy - 2) + (gpu_def->locNx) * (gpu_def->locNy) * k];
		device_test_nan(DevBuffer[i + (gpu_def->locNx)*k], __FILE__, __LINE__);
	}
}

__global__ void load_exchange_data_part_zl_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && j < (gpu_def->locNy))
	{
		DevBuffer[i + (gpu_def->locNx)*j] = DevArray[i + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy)];
		device_test_nan(DevBuffer[i + (gpu_def->locNx)*j], __FILE__, __LINE__);
	}
}

__global__ void load_exchange_data_part_zr_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && j < (gpu_def->locNy))
	{
		DevBuffer[i + (gpu_def->locNx)*j] = DevArray[i + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * (gpu_def->locNz - 2)];
		device_test_nan(DevBuffer[i + (gpu_def->locNx)*j], __FILE__, __LINE__);
	}
}

void load_exchange_data_part_xl(double* DevArray)
{
	load_exchange_data_part_xl_kernel <<< dim3(def.blocksY, def.blocksZ), dim3(BlockNY, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_xl", __FILE__, __LINE__);

	hipMemcpy(HostBuffer, DevBuffer, (def.locNy) * (def.locNz) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

void load_exchange_data_part_xr(double* DevArray)
{
	load_exchange_data_part_xr_kernel <<< dim3(def.blocksY, def.blocksZ), dim3(BlockNY, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_xr", __FILE__, __LINE__);

	hipMemcpy(HostBuffer, DevBuffer, (def.locNy) * (def.locNz) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

void load_exchange_data_part_yl(double* DevArray)
{
	load_exchange_data_part_yl_kernel <<< dim3(def.blocksX, def.blocksZ), dim3(BlockNX, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_yl", __FILE__, __LINE__);

	hipMemcpy(HostBuffer, DevBuffer, (def.locNx) * (def.locNz) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

void load_exchange_data_part_yr(double* DevArray)
{
	load_exchange_data_part_yr_kernel <<< dim3(def.blocksX, def.blocksZ), dim3(BlockNX, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_yr", __FILE__, __LINE__);

	hipMemcpy(HostBuffer, DevBuffer, (def.locNx) * (def.locNz) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

void load_exchange_data_part_zl(double* DevArray)
{
	load_exchange_data_part_zl_kernel <<< dim3(def.blocksX, def.blocksY), dim3(BlockNX, BlockNY)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_zl", __FILE__, __LINE__);
	
	hipMemcpy(HostBuffer, DevBuffer, (def.locNx) * (def.locNy) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

void load_exchange_data_part_zr(double* DevArray)
{
	load_exchange_data_part_zr_kernel <<< dim3(def.blocksX, def.blocksY), dim3(BlockNX, BlockNY)>>>(DevArray, DevBuffer);
	checkErrors("load_exchange_data_part_zr", __FILE__, __LINE__);

	hipMemcpy(HostBuffer, DevBuffer, (def.locNx) * (def.locNy) * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors("copy data to host", __FILE__, __LINE__);
}

__global__ void save_exchange_data_part_xl_kernel(double* DevArray, double* DevBuffer)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (j < gpu_def->locNy && k < (gpu_def->locNz))
	{
		DevArray[(gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k] = DevBuffer[j + (gpu_def->locNy)*k];
		device_test_nan(DevArray[(gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k], __FILE__, __LINE__);
	}
}

__global__ void save_exchange_data_part_xr_kernel(double* DevArray, double* DevBuffer)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (j < gpu_def->locNy && k < (gpu_def->locNz))
	{
		DevArray[(gpu_def->locNx) - 1 + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k] = DevBuffer[j + (gpu_def->locNy)*k];
		device_test_nan(DevArray[(gpu_def->locNx) - 1 + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * k], __FILE__, __LINE__);
	}
}

__global__ void save_exchange_data_part_yl_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && k < (gpu_def->locNz))
	{
		DevArray[i + (gpu_def->locNx) * (gpu_def->locNy) * k] = DevBuffer[i + (gpu_def->locNx)*k];
		device_test_nan(DevArray[i + (gpu_def->locNx) * (gpu_def->locNy) * k], __FILE__, __LINE__);
	}
}

__global__ void save_exchange_data_part_yr_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && k < (gpu_def->locNz))
	{
		DevArray[i + (gpu_def->locNx) * (gpu_def->locNy - 1) + (gpu_def->locNx) * (gpu_def->locNy) * k] = DevBuffer[i + (gpu_def->locNx)*k];
		device_test_nan(DevArray[i + (gpu_def->locNx) * (gpu_def->locNy - 1) + (gpu_def->locNx) * (gpu_def->locNy) * k], __FILE__, __LINE__);
	}
}

__global__ void save_exchange_data_part_zl_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && j < (gpu_def->locNy))
	{
		DevArray[i + (gpu_def->locNx) * j] = DevBuffer[i + (gpu_def->locNx)*j];
		device_test_nan(DevArray[i + (gpu_def->locNx) * j], __FILE__, __LINE__);
	}
}

__global__ void save_exchange_data_part_zr_kernel(double* DevArray, double* DevBuffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < gpu_def->locNx && j < (gpu_def->locNy))
	{
		DevArray[i + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * (gpu_def->locNz - 1)] = DevBuffer[i + (gpu_def->locNx)*j];
		device_test_nan(DevArray[i + (gpu_def->locNx) * j + (gpu_def->locNx) * (gpu_def->locNy) * (gpu_def->locNz - 1)], __FILE__, __LINE__);
	}
}

void save_exchange_data_part_xl(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNy) * (def.locNz)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_xl_kernel <<< dim3(def.blocksY, def.blocksZ), dim3(BlockNY, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_xl", __FILE__, __LINE__);
}

void save_exchange_data_part_xr(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNy) * (def.locNz)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_xr_kernel <<< dim3(def.blocksY, def.blocksZ), dim3(BlockNY, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_xr", __FILE__, __LINE__);
}

void save_exchange_data_part_yl(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNx) * (def.locNz)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_yl_kernel <<< dim3(def.blocksX, def.blocksZ), dim3(BlockNX, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_yl", __FILE__, __LINE__);
}

void save_exchange_data_part_yr(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNx) * (def.locNz)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_yr_kernel <<< dim3(def.blocksX, def.blocksZ), dim3(BlockNX, BlockNZ)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_yr", __FILE__, __LINE__);
}

void save_exchange_data_part_zl(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNx) * (def.locNy)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_zl_kernel <<< dim3(def.blocksX, def.blocksY), dim3(BlockNX, BlockNY)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_zl", __FILE__, __LINE__);
}

void save_exchange_data_part_zr(double* DevArray)
{
	hipMemcpy(DevBuffer, HostBuffer, (def.locNx) * (def.locNy)*sizeof(double), hipMemcpyHostToDevice);
	checkErrors("copy data to device", __FILE__, __LINE__);

	save_exchange_data_part_zr_kernel <<< dim3(def.blocksX, def.blocksY), dim3(BlockNX, BlockNY)>>>(DevArray, DevBuffer);
	checkErrors("save_exchange_data_part_zr", __FILE__, __LINE__);
}
